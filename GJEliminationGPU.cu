#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "common.h"

const int TILE_SIZE = 16;

__global__ void ScaleRowKernel(float* matrix, unsigned int numberOfRows, unsigned int numberOfColumns, float* outputMatrix, int current_column)
{
	int tx = (blockIdx.x*blockDim.x) + threadIdx.x;
	int ty = (blockIdx.y*blockDim.y) + threadIdx.y;
	int tID = (ty*numberOfColumns)+tx;
	if(current_column == ty && ty < numberOfRows)
	{
		if(tx < numberOfColumns)
		{
			outputMatrix[tID] = matrix[tID]/matrix[(current_column*numberOfColumns)+current_column];
		}
	}
}

__global__ void SubtractRowKernel(float* matrix, unsigned int numberOfRows, unsigned int numberOfColumns, float* outputMatrix, int current_column)
{
	int tx = (blockIdx.x*blockDim.x) + threadIdx.x;
	int ty = (blockIdx.y*blockDim.y) + threadIdx.y;
	int tID = (ty*numberOfColumns)+tx;
	if(current_column != ty && ty < numberOfRows)
	{
		if(tx < numberOfColumns)
		{
			outputMatrix[tID] = matrix[tID] - (matrix[(current_column*numberOfColumns)+tx] * matrix[(ty*numberOfColumns)+current_column]);
		}
	}
}


bool GaussianEliminationGPU( float** matrix, unsigned int numberOfRows, unsigned int numberOfColumns, float** outputMatrix, bool partialPivot )
{
	// Error return value
	hipError_t status;
	// Number of bytes in the matrix.
	int bytes = numberOfRows * numberOfColumns * sizeof(float);
	float *Md, *Pd;

	float *M = new float[bytes];
	float *P = new float[bytes];
	int count = 0;
	int rowID = 0;
	for(int i=0;i<numberOfRows;i++)
	{
		for(int j=0;j<numberOfColumns;j++)
		{
			M[count] = matrix[i][j];
			count++;
		}
	}
	// Allocate memory on the device to store each matrix
	hipMalloc((void**) &Md, bytes);
	hipMalloc((void**) &Pd, bytes);
	// Copy the host input data to the device
	hipMemcpy(Md, M, bytes, hipMemcpyHostToDevice);
	hipMemcpy(Pd, Md, bytes, hipMemcpyDeviceToDevice);
	// Specify the size of the grid and the size of the block
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid((int)ceil((float)numberOfColumns / (float)TILE_SIZE),(int)ceil((float)numberOfRows / (float)TILE_SIZE));

	//std::cout << "\nnumber of rows: "<<numberOfRows;
	for(int i=0;i<numberOfRows;i++)
	{
		//hipMemcpy(Md, M, bytes, hipMemcpyHostToDevice);

		ScaleRowKernel<<<dimGrid, dimBlock>>>(Md, numberOfRows, numberOfColumns, Pd, i);
		hipDeviceSynchronize();
		// Check for errors
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed: " << hipGetErrorString(status) << std::endl;
			hipFree(Md);
			hipFree(Pd);
			return false;
		}
		hipMemcpy(Md, Pd, bytes, hipMemcpyDeviceToDevice);
		//std::cout<<"\nscale row...";
		/*for(int j =0;j<(numberOfRows*numberOfColumns);j++)
		{
			rowID = j/numberOfColumns;
			if(rowID == i)
			{
				M[j] = P[j];
				//std::cout<<"\n"<<j<<"th element: "<<P[j];
			}
		}
		hipMemcpy(Md, M, bytes, hipMemcpyHostToDevice);*/

		SubtractRowKernel<<<dimGrid, dimBlock>>>(Md, numberOfRows, numberOfColumns, Pd, i);
		hipDeviceSynchronize();
		// Check for errors
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed: " << hipGetErrorString(status) << std::endl;
			hipFree(Md);
			hipFree(Pd);
			return false;
		}
		hipMemcpy(Md, Pd, bytes, hipMemcpyDeviceToDevice);
		//std::cout<<"\nsubtract row...";
		/*for(int j =0;j<(numberOfRows*numberOfColumns);j++)
		{
			rowID = j/numberOfColumns;
			if(rowID != i)
			{
				M[j] = P[j];
				//std::cout<<"\n"<<j<<"th element: "<<P[j];
			}
		}*/
	}
	// Retrieve the result matrix
	hipMemcpy(P, Md, bytes, hipMemcpyDeviceToHost);
	//std::cout<<"\noutput matrix GPU: ";
	/*for(int i =0;i<(numberOfColumns*numberOfRows);i++)
	{
		std::cout<<M[i]<<"\t";
	}*/
	count = 0;
	for(int i=0;i<numberOfRows;i++)
	{
		for(int j=0;j<numberOfColumns;j++)
		{
			outputMatrix[i][j] = P[count];
			count++;
		}
	}
	// Free device memory
	hipFree(Md);
	hipFree(Pd);
	delete[] P;
	delete[] M;
	// Success
	return true;
}